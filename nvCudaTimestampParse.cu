#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <time.h>

#include <hip/hip_runtime.h>

#include "customer_functions.h"
#include "cudaEGL.h"
#include "iva_metadata.h"


unsigned char bits[64];
unsigned long counter = 0L;

/**
  * Dummy custom pre-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
	return;
}

/**
  * Dummy custom post-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{


	if ( counter % 30 == 0 ) {

		unsigned long long parsedTimestamp = 0LL;

		if (sformat[0] == COLOR_FORMAT_Y8) {

			for ( int I = 0; I < 64; I++ ) {
				char * pixelPtr = (char *)sBaseAddr[0] + (100 * spitch[0]) + 100 + (4 * I);
				if ( *pixelPtr < 128 ) {
					parsedTimestamp |= (0x1LL << (63 - I));
				}
			}

			std::chrono::time_point<std::chrono::system_clock> now = std::chrono::system_clock::now();
			auto duration = now.time_since_epoch();
			unsigned long long micros = std::chrono::duration_cast<std::chrono::microseconds>(duration).count();

		}
	}

	return;
}




__global__ void retrieveTimestampBitKernel(int * pYPlanePtr, int * pUvPlanePtr, int pitch, bool * b) {

	int bitNumber = 63 - threadIdx.x;

	double ysum = 0.0;

	for ( int I = 0; I < 4; I++ ) {
		for ( int J = 0; J < 4; J++ ) {
			char * pYpixel = (char *)pYPlanePtr + ((100 + I) * pitch) + (100 + (threadIdx.x * 4) + J); 
			ysum += *pYpixel;
		}
	}


	///ysum = 1.16 * (ysum - 256.0);

	if ( ysum >= (8.0 * 255.0) ) b[bitNumber] = true;

	return;
}




static int parseTimestampOverlay(hipDeviceptr_t pYPlanePtr, hipDeviceptr_t pUvPlanePtr, int pitch, bool * b){

    dim3 threadsPerBlock(64);
    dim3 blocks(1);
    retrieveTimestampBitKernel<<<blocks,threadsPerBlock>>>((int*)pYPlanePtr, (int*)pUvPlanePtr, pitch, b);

    return 0;

}






/**
  * Performs CUDA Operations on egl image.
  *
  * @param image : EGL image
  */
static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  counter++;

  if ( (counter % 30) != 0 ) {
	return;
  }

  std::chrono::time_point<std::chrono::system_clock> nowBeforeParse = std::chrono::system_clock::now();
  auto durationBeforeParse = nowBeforeParse.time_since_epoch();
  unsigned long long microsBeforeParse = std::chrono::duration_cast<std::chrono::milliseconds>(durationBeforeParse).count();

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  bool * deviceBits;

  hipMalloc((void **)&deviceBits, 64 * sizeof(bool));
  hipMemset(deviceBits, 0, 64 * sizeof(bool));
   
  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }


  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
      parseTimestampOverlay((hipDeviceptr_t) eglFrame.frame.pPitch[0], (hipDeviceptr_t) eglFrame.frame.pPitch[1], eglFrame.pitch, deviceBits);
    } else {
      printf ("Invalid eglcolorformat\n");
    }
  } else {
      printf("Invalid frame type!!\n");
  }

  {
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  bool hostBits[64];
  hipMemcpy(&hostBits[0], deviceBits, 64*sizeof(bool), hipMemcpyDeviceToHost);

  hipFree(deviceBits);

  unsigned long long ts = 0LL;
  for ( int I = 0; I < 64; I++ ) {
	if ( ! hostBits[I] ) {
		ts |= ( 0x1LL << I );
	}
  }

  printf("HUH Thirty frames: %llu %llu %lld\n", microsBeforeParse, ts, (long long)(microsBeforeParse - ts));

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }


}

extern "C" void
init (CustomerFunction * pFuncs)
{
  //pFuncs->fPreProcess = pre_process;
  pFuncs->fPreProcess = NULL;
  pFuncs->fGPUProcess = gpu_process;
  //pFuncs->fPostProcess = post_process;
  pFuncs->fPostProcess = NULL;
  printf("libnvcuda_timestamp_overlay.so::init(): The video timestamp processing library has been initialized.\n");
}

extern "C" void
deinit (void)
{
  /* deinitialization */
}
